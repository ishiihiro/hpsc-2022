
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

__global__ void bucket_sort(int *key, int *bucket, int range){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    bucket[i%range] = 0;
    __syncthreads();
    atomicAdd(&bucket[key[i]], 1);
    extern __shared__ int scan[];
    for(int j=1; j<range; j<<=1){
        if(i<range) scan[i] = bucket[i];
        __syncthreads();
        if(i>=j && i<range) bucket[i] += scan[i-j];
        __syncthreads();
    }
    for(int j=0; j<range; j++){
        if(i < bucket[j]){
            key[i] = j;
            return;
        }
    }
}

int main() {
  int n = 50;
  int range = 5;
  int *key, *bucket;
  hipMallocManaged(&key, n*sizeof(int));
  hipMallocManaged(&bucket, range*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  bucket_sort<<<1, n, range*sizeof(int)>>>(key, bucket, range);
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
  hipFree(key);
  hipFree(bucket);
}
